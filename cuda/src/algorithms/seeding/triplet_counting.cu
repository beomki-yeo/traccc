#include "hip/hip_runtime.h"
/** TRACCC library, part of the ACTS project (R&D line)
 * 
 * (c) 2021 CERN for the benefit of the ACTS project
 * 
 * Mozilla Public License Version 2.0
 */

#include <cuda/algorithms/seeding/triplet_counting.cuh>
#include <cuda/utils/definitions.hpp>

namespace traccc{    
namespace cuda{

__global__
void triplet_counting_kernel(const seedfinder_config config,
			     const seedfilter_config filter_config,
			     internal_spacepoint_container_view internal_sp_view,
			     doublet_counter_container_view doublet_counter_view,
			     doublet_container_view mid_bot_doublet_view,
			     doublet_container_view mid_top_doublet_view,
			     triplet_counter_container_view triplet_counter_view);
    
void triplet_counting(const seedfinder_config& config,
		      const seedfilter_config& filter_config,
		      host_internal_spacepoint_container& internal_sp_container,
		      host_doublet_counter_container& doublet_counter_container,
		      host_doublet_container& mid_bot_doublet_container,
		      host_doublet_container& mid_top_doublet_container,
		      host_triplet_counter_container& triplet_counter_container,
		      vecmem::memory_resource* resource){
    
    auto internal_sp_view = get_data(internal_sp_container, resource);
    auto doublet_counter_container_view = get_data(doublet_counter_container, resource);
    auto mid_bot_doublet_view = get_data(mid_bot_doublet_container, resource);
    auto mid_top_doublet_view = get_data(mid_top_doublet_container, resource);
    auto triplet_counter_container_view = get_data(triplet_counter_container, resource);
    
    unsigned int num_threads = WARP_SIZE*6; 
    unsigned int num_blocks = internal_sp_view.headers.m_size;
    
    triplet_counting_kernel<<< num_blocks, num_threads >>>(config,
							   filter_config,
							   internal_sp_view,
							   doublet_counter_container_view,
							   mid_bot_doublet_view,
							   mid_top_doublet_view,
							   triplet_counter_container_view);
    
    CUDA_ERROR_CHECK(hipGetLastError());
    CUDA_ERROR_CHECK(hipDeviceSynchronize());	            
}

__global__
void triplet_counting_kernel(const seedfinder_config config,
			     const seedfilter_config filter_config,
			     internal_spacepoint_container_view internal_sp_view,
			     doublet_counter_container_view doublet_counter_view,
			     doublet_container_view mid_bot_doublet_view,
			     doublet_container_view mid_top_doublet_view,
			     triplet_counter_container_view triplet_counter_view){

    device_internal_spacepoint_container internal_sp_device({internal_sp_view.headers, internal_sp_view.items});
    device_doublet_counter_container doublet_counter_device({doublet_counter_view.headers, doublet_counter_view.items});	
    device_doublet_container mid_bot_doublet_device({mid_bot_doublet_view.headers, mid_bot_doublet_view.items});
    device_doublet_container mid_top_doublet_device({mid_top_doublet_view.headers, mid_top_doublet_view.items});
    device_triplet_counter_container triplet_counter_device({triplet_counter_view.headers, triplet_counter_view.items});

    auto bin_info = internal_sp_device.headers.at(blockIdx.x);
    auto internal_sp_per_bin = internal_sp_device.items.at(blockIdx.x);
    auto& num_compat_spM_per_bin = doublet_counter_device.headers.at(blockIdx.x);
    auto doublet_counter_per_bin = doublet_counter_device.items.at(blockIdx.x);
    auto num_mid_bot_doublets_per_bin = mid_bot_doublet_device.headers.at(blockIdx.x);
    auto mid_bot_doublets_per_bin = mid_bot_doublet_device.items.at(blockIdx.x);
    auto num_mid_top_doublets_per_bin = mid_top_doublet_device.headers.at(blockIdx.x);
    auto mid_top_doublets_per_bin = mid_top_doublet_device.items.at(blockIdx.x);
    auto& num_compat_mb_per_bin = triplet_counter_device.headers.at(blockIdx.x);
    auto triplet_counter_per_bin = triplet_counter_device.items.at(blockIdx.x);    
    
    size_t n_iter = num_mid_bot_doublets_per_bin/blockDim.x + 1;

    num_compat_mb_per_bin = 0;

    __syncthreads();
    
    for (size_t i_it = 0; i_it < n_iter; ++i_it){
	auto mb_idx = i_it*blockDim.x + threadIdx.x;
	auto mid_bot_doublet = mid_bot_doublets_per_bin[mb_idx];
	
	if (mb_idx >= num_mid_bot_doublets_per_bin){
	    continue;
	}

	triplet_counter_per_bin[mb_idx].mid_bot_doublet = mid_bot_doublet;
	triplet_counter_per_bin[mb_idx].n_triplets = 0;

	size_t num_triplets_per_mid_bot = 0;
	auto spM_idx = mid_bot_doublet.sp1.sp_idx;
	auto spM = internal_sp_per_bin[spM_idx];
	auto lb = mid_bot_doublet.lin;
	
	scalar iSinTheta2 = 1 + lb.cotTheta * lb.cotTheta;
	scalar scatteringInRegion2 = config.maxScatteringAngle2 * iSinTheta2;
	scatteringInRegion2 *= config.sigmaScattering * config.sigmaScattering;
	scalar curvature, impact_parameter;	
	
	size_t mb_end_idx = 0;
	size_t mt_start_idx = 0;
	size_t mt_end_idx = 0;
	
	for (int i=0; i<internal_sp_per_bin.size(); ++i){
	    if (doublet_counter_per_bin[i].n_mid_bot == 0 ||
		doublet_counter_per_bin[i].n_mid_top == 0){
		continue;
	    }
	    	    
	    mb_end_idx += doublet_counter_per_bin[i].n_mid_bot;
	    mt_end_idx += doublet_counter_per_bin[i].n_mid_top;
	    
	    if (mb_end_idx > mb_idx){
		break;
	    }
	    mt_start_idx += doublet_counter_per_bin[i].n_mid_top;
	}

	if (mt_end_idx >= mid_top_doublets_per_bin.size()){
	    mt_end_idx = fmin(mid_top_doublets_per_bin.size(), mt_end_idx);
	}	    
	
	if (mt_start_idx >= mid_top_doublets_per_bin.size()){
	    continue;
	}

	// iterate over mid-top doublets	
	for (auto mt_it = mid_top_doublets_per_bin.begin()+mt_start_idx;
	     mt_it!= mid_top_doublets_per_bin.begin()+mt_end_idx ;
	     mt_it++){
	    
	    auto lt = (*mt_it).lin;
	    
	    if (triplet_finding_helper::isCompatible(spM, lb, lt, config,
						      iSinTheta2, scatteringInRegion2,
						      curvature, impact_parameter)){
		triplet_counter_per_bin[mb_idx].n_triplets++;
	    }	    
	}

	if (triplet_counter_per_bin[mb_idx].n_triplets > 0){
	    atomicAdd(&num_compat_mb_per_bin,1);
	}	
    }    
}
    
}// namespace cuda
}// namespace traccc
