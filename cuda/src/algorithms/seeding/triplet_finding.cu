#include "hip/hip_runtime.h"
/** TRACCC library, part of the ACTS project (R&D line)
 * 
 * (c) 2021 CERN for the benefit of the ACTS project
 * 
 * Mozilla Public License Version 2.0
 */

#include <cuda/algorithms/seeding/triplet_finding.cuh>
#include <cuda/utils/definitions.hpp>

namespace traccc{    
namespace cuda{

__global__
void triplet_finding_kernel(const seedfinder_config config,
			    const seedfilter_config filter_config,
			    internal_spacepoint_container_view internal_sp_view,
			    doublet_container_view mid_bot_doublet_view,
			    doublet_container_view mid_top_doublet_view,
			    vecmem::data::jagged_vector_view< size_t > n_mb_per_spM_view,
			    vecmem::data::jagged_vector_view< size_t > n_mt_per_spM_view,
			    triplet_container_view triplet_view);    

    
void triplet_finding(const seedfinder_config& config,
		     const seedfilter_config& filter_config,
		     host_internal_spacepoint_container& internal_sp_container,
		     host_doublet_container& mid_bot_doublet_container,
		     host_doublet_container& mid_top_doublet_container,
		     vecmem::jagged_vector< size_t >& n_mb_per_spM,  
		     vecmem::jagged_vector< size_t >& n_mt_per_spM,
		     host_triplet_container& triplet_container,
		     vecmem::memory_resource* resource){

    auto internal_sp_view = get_data(internal_sp_container, resource);
    auto mid_bot_doublet_view = get_data(mid_bot_doublet_container, resource);
    auto mid_top_doublet_view = get_data(mid_top_doublet_container, resource);
    auto n_mb_per_spM_view = vecmem::get_data(n_mb_per_spM, resource);
    auto n_mt_per_spM_view = vecmem::get_data(n_mt_per_spM, resource);
    
    auto triplet_view = get_data(triplet_container, resource);
    
    unsigned int num_threads = WARP_SIZE*2; 
    unsigned int num_blocks = internal_sp_view.headers.m_size;

    triplet_finding_kernel<<< num_blocks, num_threads >>>(config,
							  filter_config,
							  internal_sp_view,
							  mid_bot_doublet_view,
							  mid_top_doublet_view,
							  n_mb_per_spM_view,
							  n_mt_per_spM_view,
							  triplet_view);
    
    CUDA_ERROR_CHECK(hipGetLastError());
    CUDA_ERROR_CHECK(hipDeviceSynchronize());	            
}

__global__
void triplet_finding_kernel(const seedfinder_config config,
			    const seedfilter_config filter_config,
			    internal_spacepoint_container_view internal_sp_view,
			    doublet_container_view mid_bot_doublet_view,
			    doublet_container_view mid_top_doublet_view,	   
			    vecmem::data::jagged_vector_view< size_t > n_mb_per_spM_view,
			    vecmem::data::jagged_vector_view< size_t > n_mt_per_spM_view,
			    triplet_container_view triplet_view){

    device_internal_spacepoint_container internal_sp_device({internal_sp_view.headers, internal_sp_view.items});
    device_doublet_container mid_bot_doublet_device({mid_bot_doublet_view.headers, mid_bot_doublet_view.items});
    device_doublet_container mid_top_doublet_device({mid_top_doublet_view.headers, mid_top_doublet_view.items});
    vecmem::jagged_device_vector< size_t > n_mb_per_spM_device(n_mb_per_spM_view);
    vecmem::jagged_device_vector< size_t > n_mt_per_spM_device(n_mt_per_spM_view);
    
    device_triplet_container triplet_device({triplet_view.headers, triplet_view.items});
    
    auto bin_info = internal_sp_device.headers.at(blockIdx.x);
    auto internal_sp_per_bin = internal_sp_device.items.at(blockIdx.x);
    auto num_mid_bot_doublets_per_bin = mid_bot_doublet_device.headers.at(blockIdx.x);
    auto mid_bot_doublets_per_bin = mid_bot_doublet_device.items.at(blockIdx.x);
    auto num_mid_top_doublets_per_bin = mid_top_doublet_device.headers.at(blockIdx.x);
    auto mid_top_doublets_per_bin = mid_top_doublet_device.items.at(blockIdx.x);
   
    size_t n_iter = num_mid_bot_doublets_per_bin/blockDim.x + 1;

    auto& num_triplets_per_bin = triplet_device.headers.at(blockIdx.x);
    
    auto triplets_per_bin = triplet_device.items.at(blockIdx.x);

    auto n_mb_per_spM = n_mb_per_spM_device.at(blockIdx.x);
    auto n_mt_per_spM = n_mt_per_spM_device.at(blockIdx.x);
    
    for (size_t i_it = 0; i_it < n_iter; ++i_it){
	auto mb_idx = i_it*blockDim.x + threadIdx.x;
	auto mid_bot_doublet = mid_bot_doublets_per_bin[mb_idx];
	
	if (mb_idx >= num_mid_bot_doublets_per_bin){
	    continue;
	}

	if (n_mb_per_spM.size() == 0 || n_mt_per_spM.size() == 0){
	    continue;
	}

	size_t num_triplets_per_mid_bot = 0;
	auto spM = internal_sp_per_bin[mid_bot_doublet.sp1.sp_idx];
	auto lb = mid_bot_doublet.lin;

	scalar iSinTheta2 = 1 + lb.cotTheta * lb.cotTheta;
	scalar scatteringInRegion2 = config.maxScatteringAngle2 * iSinTheta2;
	scatteringInRegion2 *= config.sigmaScattering * config.sigmaScattering;
	scalar curvature, impact_parameter;	
	

	size_t mb_end_idx = 0;
	size_t mt_start_idx = 0;
	size_t mt_end_idx = 0;	
	
	for (int i=0; i<n_mb_per_spM.size(); ++i){
	    auto n_mb = n_mb_per_spM[i];
	    auto n_mt = n_mt_per_spM[i];
	    
	    mb_end_idx += n_mb;
	    mt_end_idx += n_mt;
	    
	    if (mb_end_idx > mb_idx){
		break;
	    }
	    mt_start_idx += n_mt;	    

	}
	
	/*
	for (auto n_mb : n_mb_per_spM){
	    //auto n_mid_top = n_doublets.second;
	    
	    end_idx += n_doublets;	    
	    if (end_idx > mb_idx){
		break;
	    }
	    start_idx += n_doublets;	    
	}        
	*/
	
	// iterate over mid-top doublets
	for (auto mt_it = mid_top_doublets_per_bin.begin()+mt_start_idx;
	     mt_it!= mid_top_doublets_per_bin.begin()+mt_end_idx ;
	     mt_it++){

	    auto lt = (*mt_it).lin;
	    
	    if (!triplet_finding_helper::isCompatible(spM, lb, lt, config,
						      iSinTheta2, scatteringInRegion2,
						      curvature, impact_parameter)){
		continue;
	    }

	    num_triplets_per_mid_bot++;
	    auto pos = atomicAdd(&num_triplets_per_bin,1);
	    if (pos>=triplets_per_bin.size()){
		num_triplets_per_bin = triplets_per_bin.size();
		continue;
	    }
	    
	    triplets_per_bin[pos] = triplet({mid_bot_doublet.sp2,
					     mid_bot_doublet.sp1,
					     (*mt_it).sp2,
					     curvature,
					     impact_parameter,
					     -impact_parameter*filter_config.impactWeightFactor,
					     lb.Zo
		});
	}
    }
    __syncthreads();
}
    
}// namespace cuda
}// namespace traccc

