#include "hip/hip_runtime.h"
/** TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2021 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/sort.h>

#include <algorithm>
#include <cuda/algorithms/seeding/seed_selecting.cuh>
#include <cuda/utils/cuda_helper.cuh>

namespace traccc {
namespace cuda {

struct triplet_weight_descending
    : public thrust::binary_function<triplet, triplet, bool> {
    __device__ bool operator()(const triplet& lhs, const triplet& rhs) const {
        if (lhs.weight != rhs.weight) {
            return lhs.weight > rhs.weight;
        } else {
            return fabs(lhs.z_vertex) < fabs(rhs.z_vertex);
        }
    }
};

__device__ static bool triplet_weight_compare(const triplet& lhs,
                                              const triplet& rhs) {
    if (lhs.weight != rhs.weight) {
        return lhs.weight < rhs.weight;
    } else {
        return fabs(lhs.z_vertex) > fabs(rhs.z_vertex);
    }
}

__global__ void seed_selecting_kernel(
    const seedfilter_config filter_config,
    internal_spacepoint_container_view internal_sp_view,
    doublet_counter_container_view doublet_counter_view,
    triplet_counter_container_view triplet_counter_view,
    triplet_container_view triplet_view, seed_container_view seed_view);

void seed_selecting(const seedfilter_config& filter_config,
                    host_internal_spacepoint_container& internal_sp_container,
                    host_doublet_counter_container& doublet_counter_container,
                    host_triplet_counter_container& triplet_counter_container,
                    host_triplet_container& triplet_container,
                    host_seed_container& seed_container,
                    vecmem::memory_resource* resource) {
    auto internal_sp_view = get_data(internal_sp_container, resource);

    auto doublet_counter_container_view =
        get_data(doublet_counter_container, resource);
    auto triplet_counter_container_view =
        get_data(triplet_counter_container, resource);
    auto triplet_container_view = get_data(triplet_container, resource);
    auto seed_container_view = get_data(seed_container, resource);

    unsigned int num_threads = WARP_SIZE * 2;
    unsigned int num_blocks = 0;
    for (size_t i=0; i<internal_sp_view.headers.m_size; ++i){
	num_blocks += triplet_counter_container.headers[i] / num_threads +1;
    }
    
    unsigned int sh_mem =
        sizeof(triplet) * num_threads * filter_config.max_triplets_per_spM;

    seed_selecting_kernel<<<num_blocks, num_threads, sh_mem>>>(
        filter_config, internal_sp_view, doublet_counter_container_view,
        triplet_counter_container_view, triplet_container_view,
        seed_container_view);

    CUDA_ERROR_CHECK(hipGetLastError());
    CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

__global__ void seed_selecting_kernel(
    const seedfilter_config filter_config,
    internal_spacepoint_container_view internal_sp_view,
    doublet_counter_container_view doublet_counter_view,
    triplet_counter_container_view triplet_counter_view,
    triplet_container_view triplet_view, seed_container_view seed_view) {
    device_internal_spacepoint_container internal_sp_device(
        {internal_sp_view.headers, internal_sp_view.items});
    device_doublet_counter_container doublet_counter_device(
        {doublet_counter_view.headers, doublet_counter_view.items});
    device_triplet_counter_container triplet_counter_device(
        {triplet_counter_view.headers, triplet_counter_view.items});
    device_triplet_container triplet_device(
        {triplet_view.headers, triplet_view.items});
    device_seed_container seed_device({seed_view.headers, seed_view.items});

    unsigned int n_bins = internal_sp_device.headers.size();
    unsigned int bin_idx = 0;
    unsigned int ref_block_idx = 0;

    cuda_helper::get_bin_idx(n_bins,
			     triplet_counter_device,
			     bin_idx,
			     ref_block_idx);
    
    auto internal_sp_per_bin = internal_sp_device.items.at(bin_idx);
    auto& num_compat_spM_per_bin =
        doublet_counter_device.headers.at(bin_idx);
    auto doublet_counter_per_bin = doublet_counter_device.items.at(bin_idx);

    auto& num_compat_mb_per_bin = triplet_counter_device.headers.at(bin_idx);
    auto triplet_counter_per_bin = triplet_counter_device.items.at(bin_idx);

    auto& num_triplets_per_bin = triplet_device.headers.at(bin_idx);
    auto triplets_per_bin = triplet_device.items.at(bin_idx);

    auto& num_seeds = seed_device.headers.at(0);
    auto seeds = seed_device.items.at(0);

    extern __shared__ triplet triplets_per_spM[];
    __syncthreads();

    auto gid = (blockIdx.x - ref_block_idx) * blockDim.x + threadIdx.x;
    
    if (gid >= num_compat_spM_per_bin) {
	return;
    }
    
    auto& spM_loc = doublet_counter_per_bin[gid].spM;
    auto& spM_idx = spM_loc.sp_idx;
    auto& spM = internal_sp_per_bin[spM_idx];
    
    if (spM_idx >= doublet_counter_per_bin.size()) {
	return;
    }
    
    size_t n_triplets_per_spM = 0;
    
    size_t stride = threadIdx.x * filter_config.max_triplets_per_spM;
    
    for (size_t i = 0; i < num_triplets_per_bin; ++i) {
	auto& aTriplet = triplets_per_bin[i];
	auto& spB_loc = aTriplet.sp1;
	auto& spT_loc = aTriplet.sp3;
	auto& spB =
	    internal_sp_device.items[spB_loc.bin_idx][spB_loc.sp_idx];
	auto& spT =
	    internal_sp_device.items[spT_loc.bin_idx][spT_loc.sp_idx];
	
	if (spM_loc == aTriplet.sp2) {
	    seed_selecting_helper::seed_weight(filter_config, spB, spT,
					       aTriplet.weight);
	    
	    if (!seed_selecting_helper::single_seed_cut(filter_config, spB,
							aTriplet.weight)) {
		continue;
	    }
	    
	    if (n_triplets_per_spM >= filter_config.max_triplets_per_spM) {
		int begin_idx = stride;
		int end_idx = stride + filter_config.max_triplets_per_spM;
		
		// Note: min_index method gives a result different
		//       from sorting method when there are the cases where
		//       weight & z_vertex are same.
		//
		//       So min_index method reduces seed matching ratio
		//       since the cpu version is using sorting method.
		//
		//       But that doesn't mean min_index method
		//       is wrong of course
		//
		//       Let's not be so obsessed about achieving
		//       perfectly same result :))))))))
		
		int min_index =
		    std::min_element(triplets_per_spM + begin_idx,
				     triplets_per_spM + end_idx,
				     triplet_weight_compare) -
		    triplets_per_spM;
		
		auto& min_weight = triplets_per_spM[min_index].weight;
		
		if (aTriplet.weight > min_weight) {
		    triplets_per_spM[min_index] = aTriplet;
		}
		
		// (deprecated) sorting method -> good for seed matching ratio but slow
		/*
		  thrust::sort(thrust::seq,
		  triplets_per_spM+begin_idx,
		  triplets_per_spM+end_idx,
		  triplet_weight_descending());
		  
		  if (aTriplet.weight >= triplets_per_spM[end_idx-1].weight){
		  triplets_per_spM[end_idx-1] = aTriplet;
		  }
		*/
	    }
	    
	    else if (n_triplets_per_spM <
		     filter_config.max_triplets_per_spM) {
		triplets_per_spM[stride + n_triplets_per_spM] = aTriplet;
		n_triplets_per_spM++;
	    }
	}
    }
    
    // sort the triplets per spM
    // sequential version of thrust sorting algorithm is used
    thrust::sort(thrust::seq, triplets_per_spM + stride,
		 triplets_per_spM + stride + n_triplets_per_spM,
		 triplet_weight_descending());
    
    size_t n_seeds_per_spM = 0;
    
    for (size_t i = stride; i < stride + n_triplets_per_spM; ++i) {
	auto& aTriplet = triplets_per_spM[i];
	auto& spB_loc = aTriplet.sp1;
	auto& spT_loc = aTriplet.sp3;
	auto& spB =
	    internal_sp_device.items[spB_loc.bin_idx][spB_loc.sp_idx];
	auto& spT =
	    internal_sp_device.items[spT_loc.bin_idx][spT_loc.sp_idx];
	
	if (n_seeds_per_spM >= filter_config.maxSeedsPerSpM + 1) {
	    break;
	}
	
	if (seed_selecting_helper::cut_per_middle_sp(filter_config, spB,
						     aTriplet.weight) ||
	    n_seeds_per_spM == 0) {
	    auto pos = atomicAdd(&num_seeds, 1);
	    
	    if (pos >= seeds.size()) {
		break;
	    }
	    n_seeds_per_spM++;
	    
	    seeds[pos] = seed({spB.m_sp, spM.m_sp, spT.m_sp,
			       aTriplet.weight, aTriplet.z_vertex});
	}
    }
}

}  // namespace cuda
}  // namespace traccc
